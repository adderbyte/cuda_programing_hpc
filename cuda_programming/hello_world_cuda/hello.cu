

#include <hip/hip_runtime.h>
#include<stdio.h>
//  __global__ is the global kernel specifier.
// identifies a kernel function (cuda c functions)
__global__ void cuda_hello(){
    // to print 
    printf("Hello World from GPU!\n");
}

int main() {
    // <<<  >>> is the execution configuration syntax
    // it specify the number of threads that will
    //	execute  the cuda_hello program 	
    cuda_hello<<<1,1>>>();

   // synchronize execution
   // see discussion in the link
   // "https://stackoverflow.com/questions/19193468/why-do-we-need-cudadevicesynchronize-in-kernels-with-device-printf"
    hipDeviceSynchronize(); 

    return 0;
}
