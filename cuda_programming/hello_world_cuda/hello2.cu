
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    // this time print the thread index 
  // for simplicity print only for thread index equals 1
    if (threadIdx.x == 1 ){
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
		}
  // note use of threadIdx.x and blockIdx.x to get
  // thread and block index respectively
}

int main() {
    // specify the number of threads 
    print_kernel<<<10, 10>>>();
    // synchronize execution between host and device
    hipDeviceSynchronize();
    return 0;
	
}
