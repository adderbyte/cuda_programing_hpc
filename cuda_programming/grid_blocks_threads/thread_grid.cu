
#include <hip/hip_runtime.h>
#include <stdio.h>
  
__global__ void kernelA(){
    // Giant conditional so that it only prints once, this would not be done in pactice
    if (blockIdx.x == 0 & blockIdx.y == 1 & blockIdx.z == 0 & threadIdx.x == 1 & threadIdx.y == 0 & threadIdx.z == 1) {
        printf("gridDim   (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
        printf("blockDim  (%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
        printf("blockIdx  (%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z);
        printf("threadIdx (%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);
        // minimum unit being executed by compute engine at the same time 
	// called wave front in AMD. It is not set by the programmer 
	printf("warpSize  (%d)\n", warpSize);

    }
}
  
int main()
{
    hipSetDevice(0);
         
    // dim3 is an integer vector type
    dim3 blocks(50, 100, 50);
    dim3 threads(8, 8, 16);
    kernelA <<<blocks,threads>>>();
  
    hipDeviceSynchronize();
     
    hipDeviceReset();
      
    return 0;
}
