#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <math.h>
#include <iostream>
 

 
/************************************************
   For more examples visit:
   https://github.com/thrust/thrust/tree/master/examples

 ************************************************/





int main(void)
{  
   
  // H has storage for 4 integers
  thrust::host_vector<float> H(4);

  // initialize individual elements
  for(int i = 0; i < H.size(); i++)
  {
	  
   H[i]  =log(i+0.004)*cos(i);
   
  // std::cout <<H[i]  << std::endl; 
  }

 
  // H.size() returns the size of vector H
  std::cout << "H has size " << H.size() << std::endl;

  // print contents of H
  for(int i = 0; i < H.size(); i++)
  {
    std::cout << "H[" << i << "] = " << H[i] << std::endl;
  }

  // resize H
  H.resize(2);
    
  std::cout << "H now has size " << H.size() << std::endl;

  // Copy host_vector H to device_vector D
  thrust::device_vector<float> D = H;
    
  // elements of D can be modified
  for(int i = 0; i < D.size(); i++)
  {D[i]  = H[i]*cos(i)*M_PI;
   // std::cout <<H[i]  << std::endl; 
  }


  // print contents of D
  for(int i = 0; i < D.size(); i++)
  {
    std::cout << "D[" << i << "] = " << D[i] << std::endl;
  }

  // H and D are automatically destroyed when the function returns
  return 0;
}
