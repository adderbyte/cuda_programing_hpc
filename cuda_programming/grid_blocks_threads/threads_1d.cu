
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void print_kernel() {

    
    printf("Hello from block %d, threadInd x %d,threadInd  y %d,threadInd z  %d ,blockDim x %d, blockDim y %d,blockDim z %d \n", blockIdx.x, threadIdx.x,threadIdx.y,threadIdx.z,blockDim.x,blockDim.y,blockDim.z );
                

}

int main() {

    // specify Number of Blocks and threadPerBlock
    // 2 is the block size --we assume a 1d grid . 4 is the number of threads.
    // the blockDim.y should return 1 but 4 for x direction since this is 1d grid
    print_kernel<<<2,4>>>();

    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();

    // Destroys and cleans up all resources associated with the current device.
    // It will reset the device immediately. It is the caller's responsibility
    //    to ensure that the device work has completed
    hipDeviceReset();
    return 0;
}

