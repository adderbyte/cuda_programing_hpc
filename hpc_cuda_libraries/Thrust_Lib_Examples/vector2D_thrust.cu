#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <math.h>
#include <iostream> 
#if !defined(ARRAY_SIZE)
    #define ARRAY_SIZE(x) (sizeof((x)) / sizeof((x)[0]))
#endif
 
/************************************************
  
   Examples here are based on the quick guide 
   https://github.com/thrust/thrust/tree/master/examples

 ************************************************/

int main(void)
{  
   
  // H is a vector 3 by 3 
  int  n = 6;
  int m = 4 ;
  
 
  thrust::host_vector<float> vectors[n];
  // initialise the elements of thrust
  for(int i = 0; i < n ; i++)
  {
  vectors[i] = thrust::host_vector<float>(m);

  }
  
  // initialise  device vector
  thrust::device_vector<float> device_vectors[n] ;
  
 // copy host vector into device vector 
  for(int i = 0; i < n ; i++)
  {
  device_vectors[i] = vectors[i];
  
  }
  //finished copying    
  std::cout <<"Finished copying host to device ... " << std::endl; 
  
  // size of each device vector  
  std::cout << "There are "<< n <<" device vectors"<<".Each of size:  " <<  device_vectors[1].size() << std::endl; 


  //resizing the device vector . Do this for the n vectors
   for(int i = 0; i < n ; i++)
  {
  device_vectors[i].resize(10);
  
  }
  std::cout <<"Resizing device vector ... .\nEach Device vector is now of size:  " <<  device_vectors[1].size() << std::endl; 
  
  // another way to go about this allocation is using the new operator
   
  std::cout <<"Another array allocation strategy." << std::endl ; 
  thrust::device_vector<float>* array_pointer = NULL;   // Pointer to device vector, initialize to nothing.
  array_pointer = new thrust::device_vector<float>[n];  // Allocate n device vector and save ptr in a.
  
  // Use a as a normal array
   for(int i = 0; i < n ; i++)
  {
  
  // copy host to device. just same as the previous copying 	  
   array_pointer[i] = vectors[i];
   for(int j =0 ; j < m ; j++)
   {
    // assign each  device vectors some values	   
    array_pointer[i][j] = cos(j) * M_PI * cos(i)   ;
    
   std::cout <<"device_vector["<< i << "]["<< j << "]="  << array_pointer[i][j] << std::endl ; 
   }
   //std::cout<<std::endl ; 
   }

   delete [] array_pointer;  // When done, free memory pointed to by array_pointer.
  array_pointer = NULL;  
  
 return 0;
}



   
