
#include <hip/hip_runtime.h>
#include <stdio.h>
  
__global__ void kernelA(int N){
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
     
    // Conditional statement to exit if index (globalThreadId) is out of bounds
    if(globalThreadId >= N) {
        return;
    }
 
    //Insert code here
    printf("Hello from block %d, threadInd x %d,threadInd  y %d,threadInd z  %d ,blockDim x %d, blockDim y %d,blockDim z %d \n", blockIdx.x, threadIdx.x,threadIdx.y,threadIdx.z,blockDim.x,blockDim.y,blockDim.z );
}
  
int main()
{
    // More realistic GPU problem size
    int problemSize = 4; // try with 1000 or 100000000
    //set the device on which the host execute files 
    hipSetDevice(0);
     
    // On average a good thread count, the best thread count varies based on the situation
    int threadCount = 2; // try with 256 which is the averagely good size
    // Simple way to ensure enough threads are launched
    //    may result in launching more threads than needed though
    int blockCount = ceil(problemSize/threadCount);
     
    kernelA <<<blockCount, threadCount>>>(problemSize);
  
    hipDeviceSynchronize();
     
    hipDeviceReset();
      
    return 0;
}
