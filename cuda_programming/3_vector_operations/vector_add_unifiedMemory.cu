
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 100
#include <math.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {

       //to  get global thread id

       // first compute thread id
       int blockId = blockIdx.x + blockIdx.y * gridDim.x;
       // then the global thread id
       int id = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

      // make sure we dont go out of thread index	
       if( id <  N ){
        out[id] = a[id] + b[id];
    }
}

int main(){
    float *a, *b, *out; 
    
    //comment out this since we will be using unified memory
    //float *d_a,*d_b,*d_c;
    
    
    //comment out Allocate memory. Use unified memory
    
    // *******************************
    //a   = (float*)malloc(sizeof(float) * N);
    //b   = (float*)malloc(sizeof(float) * N);
    //out = (float*)malloc(sizeof(float) * N);
    //*************************************
    
    //No need to allocate memory to device
    //**********************************************
    //cudaMalloc((void**)&d_a,sizeof(float)*N );
    //cudaMalloc((void**)&d_b,sizeof(float)*N );
    //cudaMalloc((void**)&d_c,sizeof(float)*N );
    //******************************************


     // unified memory
    //*******************************************************
    hipMallocManaged(&a, N*sizeof(float));
    hipMallocManaged(&b, N*sizeof(float));
    hipMallocManaged(&out, N*sizeof(float));
    // *******************************************************
  
     // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = sin(i)*sin(i)+cos(i); b[i] = cos(i)*cos(i)+sin(i);
    }



    // transfer data from host to device
    // we use the unifies memory in this example. Thus we comment line the
    // 2 lines below . 
   
    // ****************************************************
    // cudaMemcpy(d_a, a, sizeof(float) * N, cudaMemcpyHostToDevice);
    // cudaMemcpy(d_b, b, sizeof(float) * N, cudaMemcpyHostToDevice);
    // *****************************************************
    
    // define 2d block. Note this is 1 grid but 2d block . use the cheat cheat
    // to get how to compute the global id 
    
    // to keep it simpple 2 D blocks
    dim3 threads(2,5);

    // define 2 D grid

    dim3 blocks(2,5);

    // Main function
    vector_add<<<blocks,threads>>>(out,a, b, N);
    
    // no need to copy result back
    //**********************************************************
    //copy result back to host
    //cudaMemcpy(out, d_c,sizeof(float) * N , cudaMemcpyDeviceToHost);
    //**********************************************************

    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();



    // print results
    int i;
    for (i=0;i <N;i++) {
    printf("%lf,",out[i]);  }
    
    
    //clean up after executing kernel
    hipFree(a);hipFree(b);hipFree(out);
    
    // free memory if not using unified memory
    //free(a);free(b);free(out);
}

