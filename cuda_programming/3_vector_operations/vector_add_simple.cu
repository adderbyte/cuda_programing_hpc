
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 100
#include <math.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
        
       // get global thread id	
       int id = blockIdx.x *blockDim.x + threadIdx.x;
       // make sure we dont go out of thread index	
       if( id <  N ){
        out[id] = a[id] + b[id];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a,*d_b,*d_c;
    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    //Allocate device memory for a,b,c
    hipMalloc((void**)&d_a,sizeof(float)*N );
    hipMalloc((void**)&d_b,sizeof(float)*N );
    hipMalloc((void**)&d_c,sizeof(float)*N );
    
     // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = sin(i)*sin(i)+cos(i); b[i] = cos(i)*cos(i)+sin(i);
    }



    // transfer data from host to device
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    // cudaMemcpy(d_c, out, sizeof(float) * N, cudaMemcpyHostToDevice);

    // Main function
    vector_add<<<1,100>>>(d_c,d_a, d_b, N);

    //copy result back to host
    hipMemcpy(out, d_c,sizeof(float) * N , hipMemcpyDeviceToHost);
    
    // print results
    int i;
    for (i=0;i <N;i++) {
    printf("%lf,",out[i]);  }
    
    //  synchronize execution
    //cudaDeviceSynchronize();
    //clean up after executing kernel
    hipFree(d_a);hipFree(d_b);hipFree(d_c);
    free(a);free(b);free(out);
}

